
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

__global__ void sumDev(float *a,float *b,float *c){
  int i=threadIdx.x;
  printf("Thread %d\n",i);
  c[i]=a[i]+b[i];
}

void sumHost(float *a, float *b, float *c, const int n) {
	for(int i=0;i<n;i++){
    c[i]=a[i]+b[i];
  }
}

void init(float *a,int n){
  time_t t;
  srand((unsigned int) time(&t));

  for(int i=0;i<n;i++){
    a[i]=(rand()&0xFF)/10.0f;
  }
}

int main(int argc, char **argv) {
  int n=32;
  size_t bytes=n*sizeof(float);

  float *ha,*hb,*hc;
  ha=(float *)malloc(bytes);
  hb=(float *)malloc(bytes);
  hc=(float *)malloc(bytes);

  float *da,*db,*dc;
  hipMalloc(&da, bytes);
  hipMalloc(&db, bytes);
  hipMalloc(&dc, bytes);

  init(ha,n);
  init(hb,n);

  hipMemcpy(da,ha,n,hipMemcpyHostToDevice);
  hipMemcpy(db,hb,n,hipMemcpyHostToDevice);

  sumDev<<<1,n>>>(da,db,dc);

  hipMemcpy(hc,dc,n,hipMemcpyHostToDevice);

  printf("%f+%f=%f\n",ha[0],hb[0],hc[0]);

  free(ha);
  free(hb);
  free(hc);

  hipFree(da);
  hipFree(db);
  hipFree(dc);
	return(0);
}
