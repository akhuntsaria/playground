
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU thread %d!\n",threadIdx.x);
}

int main(void)
{
	printf("Hello World from CPU!\n");

	helloFromGPU <<<1,10000>>>();
	hipDeviceReset();
}
